#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define BLOCK_SIZE 16
#define STR_SIZE 256

#define MAX_PD (3.0e6)
/* required precision in degrees */
#define PRECISION 0.001
#define SPEC_HEAT_SI 1.75e6
#define K_SI 100
/* capacitance fitting factor */
#define FACTOR_CHIP 0.5

/* chip parameters */
float t_chip = 0.0005;
float chip_height = 0.016;
float chip_width = 0.016;
/* ambient temperature, assuming no package at all */
float amb_temp = 80.0;

void fatal(const char *s)
{
  fprintf(stderr, "Error: %s\n", s);
}

void readinput(float *vect, int grid_rows, int grid_cols, int layers, char *file)
{
  int i, j, k;
  FILE *fp;
  char str[STR_SIZE];
  float val;

  if ((fp = fopen(file, "r")) == 0)
    fatal("The file was not opened");

  for (i = 0; i <= grid_rows - 1; i++)
    for (j = 0; j <= grid_cols - 1; j++)
      for (k = 0; k <= layers - 1; k++)
      {
        if (fgets(str, STR_SIZE, fp) == NULL)
          fatal("Error reading file\n");
        if (feof(fp))
          fatal("not enough lines in file");
        if ((sscanf(str, "%f", &val) != 1))
          fatal("invalid file format");
        vect[i * grid_cols + j + k * grid_rows * grid_cols] = val;
      }

  fclose(fp);
}

void writeoutput(float *vect, int grid_rows, int grid_cols, int layers, char *file)
{
  int i, j, k, index = 0;
  FILE *fp;
  char str[STR_SIZE];

  if ((fp = fopen(file, "w")) == 0)
    printf("The file was not opened\n");

  for (i = 0; i < grid_rows; i++)
    for (j = 0; j < grid_cols; j++)
      for (k = 0; k < layers; k++)
      {
        sprintf(str, "%d\t%g\n", index, vect[i * grid_cols + j + k * grid_rows * grid_cols]);
        fputs(str, fp);
        index++;
      }

  fclose(fp);
}

__global__ void computeTempGPU(float *pIn, float *tIn, float *tOut,
                               int nx, int ny, int nz, float Cap,
                               float Rx, float Ry, float Rz,
                               float dt, int numiter)
{
  extern __shared__ float shared_mem[];

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;

  int local_x = threadIdx.x;
  int local_y = threadIdx.y;
  int local_z = threadIdx.z;

  int shared_size_x = blockDim.x + 2;
  int shared_size_y = blockDim.y + 2;
  int shared_size_z = blockDim.z + 2;

  int shared_index = local_x + local_y * shared_size_x + local_z * shared_size_x * shared_size_y;

  if (x < nx && y < ny && z < nz)
  {
    int c = x + y * nx + z * nx * ny;

    float ce, cw, cn, cs, ct, cb, cc;
    float stepDivCap = dt / Cap;
    ce = cw = stepDivCap / Rx;
    cn = cs = stepDivCap / Ry;
    ct = cb = stepDivCap / Rz;

    cc = 1.0 - (2.0 * ce + 2.0 * cn + 3.0 * ct);

    for (int iter = 0; iter < numiter; ++iter)
    {
      shared_mem[shared_index] = tIn[c];

      __syncthreads();

      int w = local_x > 0 ? shared_index - 1 : c;
      int e = local_x < blockDim.x - 1 ? shared_index + 1 : c;
      int n = local_y > 0 ? shared_index - shared_size_x : c;
      int s = local_y < blockDim.y - 1 ? shared_index + shared_size_x : c;
      int b = local_z > 0 ? shared_index - shared_size_x * shared_size_y : c;
      int t = local_z < blockDim.z - 1 ? shared_index + shared_size_x * shared_size_y : c;

      tOut[c] = cc * shared_mem[shared_index] +
                cw * shared_mem[w] +
                ce * shared_mem[e] +
                cs * shared_mem[s] +
                cn * shared_mem[n] +
                cb * shared_mem[b] +
                ct * shared_mem[t] +
                (dt / Cap) * pIn[c] +
                ct * amb_temp;

      // Swap pointers for the next iteration
      float *temp = tIn;
      tIn = tOut;
      tOut = temp;

      __syncthreads();
    }
  }
}

int main(int argc, char **argv)
{
  if (argc != 7)
  {
    fprintf(stderr, "Usage: %s <rows/cols> <layers> <iterations> <powerFile> <tempFile> <outputFile>\n", argv[0]);
    exit(1);
  }

  char *pfile, *tfile, *ofile;
  int iterations = atoi(argv[3]);

  pfile = argv[4];
  tfile = argv[5];
  ofile = argv[6];
  int numCols = atoi(argv[1]);
  int numRows = atoi(argv[1]);
  int layers = atoi(argv[2]);

  /* calculating parameters */

  float dx = chip_height / numRows;
  float dy = chip_width / numCols;
  float dz = t_chip / layers;

  float Cap = FACTOR_CHIP * SPEC_HEAT_SI * t_chip * dx * dy;
  float Rx = dy / (2.0 * K_SI * t_chip * dx);
  float Ry = dx / (2.0 * K_SI * t_chip * dy);
  float Rz = dz / (K_SI * dx * dy);

  float max_slope = MAX_PD / (FACTOR_CHIP * t_chip * SPEC_HEAT_SI);
  float dt = PRECISION / max_slope;

  float *powerIn, *tempOut, *tempIn;
  int size = numCols * numRows * layers;

  powerIn = (float *)calloc(size, sizeof(float));
  tempIn = (float *)calloc(size, sizeof(float));
  tempOut = (float *)calloc(size, sizeof(float));

  readinput(powerIn, numRows, numCols, layers, pfile);
  readinput(tempIn, numRows, numCols, layers, tfile);

  // Device memory
  float *d_powerIn, *d_tempIn, *d_tempOut;

  hipMalloc((void **)&d_powerIn, size * sizeof(float));
  hipMalloc((void **)&d_tempIn, size * sizeof(float));
  hipMalloc((void **)&d_tempOut, size * sizeof(float));

  // Copy input data to device
  hipMemcpy(d_powerIn, powerIn, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_tempIn, tempIn, size * sizeof(float), hipMemcpyHostToDevice);

  dim3 block_dim(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid_dim((numCols + BLOCK_SIZE - 1) / BLOCK_SIZE, (numRows + BLOCK_SIZE - 1) / BLOCK_SIZE, (layers + BLOCK_SIZE - 1) / BLOCK_SIZE);

  size_t shared_mem_size = (BLOCK_SIZE + 2) * (BLOCK_SIZE + 2) * (BLOCK_SIZE + 2) * sizeof(float);

  // Launch GPU kernel
  computeTempGPU<<<grid_dim, block_dim, shared_mem_size>>>(d_powerIn, d_tempIn, d_tempOut, numCols, numRows, layers, Cap, Rx, Ry, Rz, dt, iterations);

  // Copy result back to host
  hipMemcpy(tempOut, d_tempOut, size * sizeof(float), hipMemcpyDeviceToHost);

  // Write output to file
  writeoutput(tempOut, numRows, numCols, layers, ofile);

  // Cleanup
  hipFree(d_powerIn);
  hipFree(d_tempIn);
  hipFree(d_tempOut);
  free(powerIn);
  free(tempIn);
  free(tempOut);

  return 0;
}
